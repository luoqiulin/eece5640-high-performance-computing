#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

const int N = 64;
const int average_threads = 16;
const int blocks_number = 4;
const int opr_interspace = average_threads + 2;

__global__ void normal(float matrix_1[N][N][N], float matrix_2[N][N][N]){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i > 0 && j > 0 && k > 0 && i < N - 1 && j < N - 1 && k < N - 1){
        int tmp1 = matrix_1[i - 1][j][k] + matrix_1[i + 1][j][k] + matrix_1[i][j - 1][k];
        int tmp2 = matrix_1[i][j + 1][k] + matrix_1[i][j][k - 1] + matrix_1[i][j][k + 1];
        matrix_2[i][j][k] = 0.8 * (tmp1 + tmp2);
    }
}

__global__ void specific(float matrix_1[N][N][N], float matrix_2[N][N][N]){   
    float __shared__ public_matrix[opr_interspace][opr_interspace][opr_interspace];
    int thx = threadIdx.x + 1;
    int thy = threadIdx.y + 1;
    int thz = threadIdx.z + 1;
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    for(int r = -1; r < 2; r += 1){
        for(int s = -1; s < 2; s += 1){
            for(int t = -1; t < 2; t += 1){
                if(i + r < N && j + s < N && k + t < N){
                    public_matrix[thx + r][thy + s][thz + t] = matrix_1[i + r][j + s][k + t];
                }
            }
        }
    }
    __syncthreads();
    if (i > 0 && j > 0 && k > 0 && i < N - 1 && j < N - 1 && k < N - 1){   
        int tmp1 = public_matrix[thx - 1][thy][thz] + public_matrix[thx + 1][thy][thz] + public_matrix[thx][thy - 1][thz];
        int tmp2 = public_matrix[thx][thy + 1][thz] + public_matrix[thx][thy][thz - 1] + public_matrix[thx][thy][thz + 1];
        matrix_2[i][j][k] = 0.8 * (tmp1 + tmp2);
    }
}

int main(int argc, char *argv[]){
    float * tiled_matrix_1, * non_tiled_matrix_1, * matrix_2; 
    hipMallocManaged(&tiled_matrix_1, N * N * N * sizeof(float));
    hipMallocManaged(&non_tiled_matrix_1, N * N * N * sizeof(float));
    hipMallocManaged(&matrix_2, N * N * N * sizeof(float));
    float matrix_2_values[N][N][N];
    float matrix_1_values[N][N][N];
    memset(matrix_1_values, 0, N * N * N * sizeof(matrix_1_values[0][0][0]));
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            for(int k = 0; k < N; k++){
                matrix_2_values[i][j][k] = rand() % 100 + 1;
                matrix_2_values[i][j][k] = (float)rand() / (float)(RAND_MAX) * matrix_2_values[i][j][k];
            }
        }
    }
    float (*tiled_matrix_1_values)[N][N] = reinterpret_cast<float (*)[N][N]>(tiled_matrix_1);
    float (*non_tiled_matrix_1_values)[N][N] = reinterpret_cast<float (*)[N][N]>(non_tiled_matrix_1);
    memcpy(matrix_2, &matrix_2_values[0][0][0], sizeof(matrix_2_values));
    dim3 avgThreads_blocks(average_threads, average_threads, average_threads);
    dim3 blocks(blocks_number, blocks_number, blocks_number);
    
    hipEvent_t initial_time, end_time;
    hipEvent_t initial_time_2, end_time_2;

    hipEventCreate(&initial_time);
    hipEventCreate(&end_time);
    hipEventRecord(initial_time);
    specific<<<blocks, avgThreads_blocks>>>(reinterpret_cast<float (*)[N][N]>(matrix_2), tiled_matrix_1_values);
    hipEventRecord(end_time);
    hipEventSynchronize(end_time);
    float duration_1 = 0;
    hipEventElapsedTime(&duration_1, initial_time, end_time);
    printf("non tiled method: %f milliseconds\n", duration_1);

    hipEventCreate(&initial_time_2);
    hipEventCreate(&end_time_2);
    hipEventRecord(initial_time_2);
    normal<<<blocks, avgThreads_blocks>>>(reinterpret_cast<float (*)[N][N]>(matrix_2), non_tiled_matrix_1_values);
    hipEventRecord(end_time_2);
    hipEventSynchronize(end_time_2);
    float duration_2 = 0;
    hipEventElapsedTime(&duration_2, initial_time_2, end_time_2);
    printf("tiled method: %f milliseconds\n", duration_2);
    hipDeviceSynchronize();
    return 0;
}
