#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <iostream>
#include <limits>
#include <stdio.h>

using namespace std;

const long long int avg_threads = 16; 
const long long int blocks_number = 640;
const long long int iterations_number = 100000000; 

__global__ void leibniz_method_gpu(long long int *series_add){
	__shared__ long long int compute_arr[avg_threads];

	int series = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t cur_state;
	hiprand_init(clock64(), series, 0, &cur_state);
	compute_arr[threadIdx.x] = 0;

	for (int i = 0; i < iterations_number; i++){
		float xt = hiprand_uniform(&cur_state); 
		float yt = hiprand_uniform(&cur_state); 
		float tmp = xt * xt + yt * yt;
		compute_arr[threadIdx.x] = compute_arr[threadIdx.x] - int(tmp) + 1; 
	}
	if (threadIdx.x == 0){
		series_add[blockIdx.x] = 0;
		for (int i = 0; i < avg_threads; i++) {
			series_add[blockIdx.x] = series_add[blockIdx.x] + compute_arr[i];
		}
	}
}

int main(){	
	long long int *final_result;
	long long int *tmp_result;
	final_result = new long long int[blocks_number]; 
	hipMalloc(&tmp_result, sizeof(long long int) * blocks_number); 
	leibniz_method_gpu<<<blocks_number, avg_threads>>>(tmp_result);
	hipMemcpy(final_result, tmp_result, sizeof(long long int) * blocks_number, hipMemcpyDeviceToHost);
	hipFree(tmp_result);

	long long int series_final_add = 0;
	for (int k = 0; k < blocks_number; k++) {
		series_final_add = series_final_add + final_result[k];
	}
	long long int tests_number = blocks_number * iterations_number * avg_threads;
	cout << "Test number:" << tests_number << endl;
	printf("Results: %.16f\n", 4.0 * (float)series_final_add / (float)tests_number);
	return 0;
}
