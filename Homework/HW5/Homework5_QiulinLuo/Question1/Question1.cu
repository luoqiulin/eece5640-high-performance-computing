#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

const int bins_number = 10;
const int blocks_number = 512;
const int blocks_average_threads = 64;
const int omp_threads_number = 64;
int threads_total_number = blocks_number * blocks_average_threads;

__host__ __device__ int bins_verify(int element, float* mopr, float* Mopr){
    for(int k = 0; k < bins_number; k++) 
    {
        if(element >= mopr[k] && element < Mopr[k]){
            return k;
        }       
    }
    return -1;
}

__global__ void create_hist(int* value, int* bin_init, int* belement, float* mopr, float* Mopr, int* mtid, int* Mtid){   
    int __shared__ average_hists[blocks_average_threads][bins_number];
    int sqe_number;
    for (int i = 0; i < bins_number; i = i + 1){
        average_hists[threadIdx.x][i] = 0;
    }  

    __syncthreads();

    for(int j = mtid[blockIdx.x*blockDim.x + threadIdx.x]; j < Mtid[blockIdx.x*blockDim.x + threadIdx.x]; j++){
        sqe_number = bins_verify(value[j], mopr, Mopr);    
        average_hists[threadIdx.x][sqe_number] = average_hists[threadIdx.x][sqe_number] + 1;
        belement[sqe_number] = value[j];
    }

    __syncthreads();
    
    if (threadIdx.x == 0){
        for (int tmp1 = 0; tmp1 < bins_number; tmp1 = tmp1 + 1){   
            for (int tmp2 = 0; tmp2 < blocks_average_threads; tmp2 = tmp2 + 1) {   
                bin_init[blockIdx.x * bins_number + tmp1] += average_hists[tmp2][tmp1];
            }
        }
    }
}

__global__ void bins_increment(int* bin_init, int* bin_init_aft){   
    for (int tmp1 = 0; tmp1 < bins_number; tmp1 = tmp1 + 1){   
        for (int tmp2 = 0; tmp2 < blocks_number; tmp2 = tmp2 + 1){   
            bin_init_aft[tmp1] = bin_init_aft[tmp1] + bin_init[tmp2 * bins_number + tmp1];
        }
    }
}

void value_initialization(int* value, int value_extent){
    for(int i = 0; i < value_extent; i++){
        value[i] = rand() % 10000000 + 1;
    }
}

void value_reference(int* mtid, int* Mtid, int value_extent){
    float blank = floor((float)value_extent / (float)threads_total_number);
    for(int i = 0; i < threads_total_number; i++){
        mtid[i] = i * blank;
        Mtid[i] = (i + 1) * blank;
        if(i == threads_total_number - 1){
            Mtid[i] = value_extent;
        }
    }
}

void bins_installing(float* mopr, float* Mopr, int bins_number){
    float blank = ceil((float)(10000000 - 1) / (float)bins_number);
    for(int i = 0; i < bins_number; i++){
        mopr[i] = i * blank + 1;
        if(i == bins_number - 1){
            Mopr[i] = 10000001;
        }else{
            Mopr[i] = (i + 1) * blank + 1;
        }
    }
}

void hist_output(int* hist_value){
    int count = 0;
    for(int i = 0; i < bins_number; i++){
        count += hist_value[i];
        printf("number of values in bins %d: %d\n", i, hist_value[i]);
    }
}

void cuda_version(int* value, int value_extent, float* mopr, float* Mopr){   
    srand(time(NULL));
    struct timespec initial_1, finish_1, initial_2, finish_2;
    int* value_gpu;
    int* mtid, * Mtid;
    int* cuda_mtid, * cuda_Mtid;
    float* cuda_mopr, * cuda_Mopr;
    int* cuda_bin_init;
    int* bin_init_aft, * cuda_bin_init_aft;
    int* belement, * cuda_belement;

    clock_gettime(CLOCK_MONOTONIC, &initial_1);

    hipMalloc((void**) &cuda_mopr, sizeof(float)*bins_number);
    hipMalloc((void**) &cuda_Mopr, sizeof(float)*bins_number);
    mtid = (int*)calloc(threads_total_number, sizeof(int));
    Mtid = (int*)calloc(threads_total_number, sizeof(int));
    hipMalloc((void**) &cuda_mtid, sizeof(int)*threads_total_number);
    hipMalloc((void**) &cuda_Mtid, sizeof(int)*threads_total_number);
    hipMalloc((void**) &cuda_bin_init, sizeof(int)*bins_number*blocks_number);
    hipMemset(cuda_bin_init, 0, sizeof(int)*bins_number*blocks_number);
    bin_init_aft = (int*)calloc(bins_number, sizeof(int));
    hipMalloc((void**) &cuda_bin_init_aft, sizeof(int)*bins_number);
    hipMemset(bin_init_aft, 0, sizeof(int)*bins_number);
    belement = (int*)calloc(bins_number, sizeof(int));
    hipMalloc((void**) &cuda_belement, sizeof(int)*bins_number);
    value_reference(mtid, Mtid, value_extent);
    hipMalloc((void**) &value_gpu, sizeof(int)*value_extent);
    hipMemcpy(value_gpu, value, sizeof(int)*value_extent, hipMemcpyHostToDevice);   
    hipMemcpy(cuda_mopr, mopr, sizeof(int)*bins_number, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Mopr, Mopr, sizeof(int)*bins_number, hipMemcpyHostToDevice);
    hipMemcpy(cuda_mtid, mtid, sizeof(int)*threads_total_number, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Mtid, Mtid, sizeof(int)*threads_total_number, hipMemcpyHostToDevice);

    clock_gettime(CLOCK_MONOTONIC, &initial_2);
    create_hist<<<blocks_number, blocks_average_threads>>>(value_gpu, cuda_bin_init, cuda_belement, cuda_mopr, cuda_mopr, cuda_mtid, cuda_mtid);
    bins_increment<<<1, 1>>>(cuda_bin_init, cuda_bin_init_aft);
    clock_gettime(CLOCK_MONOTONIC, &finish_2);

    double duration_2 = (finish_2.tv_sec - initial_2.tv_sec);
    duration_2 += (finish_2.tv_nsec - initial_2.tv_nsec) / 1000000000.0;

    hipMemcpy(bin_init_aft, cuda_bin_init_aft, sizeof(int)*bins_number, hipMemcpyDeviceToHost);
    hipMemcpy(belement, cuda_belement, sizeof(int)*bins_number, hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC, &finish_1);
    double duration_1 = (finish_1.tv_sec - initial_1.tv_sec);
    duration_1 += (finish_1.tv_nsec - initial_1.tv_nsec) / 1000000000.0;

    printf("GPU Time: %f\n", duration_1);
    //hist_output(bin_init_aft);
    hipFree(value_gpu); 
    hipFree(cuda_mopr);
    hipFree(cuda_Mopr);
    hipFree(cuda_mtid);
    hipFree(cuda_Mtid);
    hipFree(cuda_bin_init);
    hipFree(cuda_bin_init_aft);
    hipFree(cuda_belement);
}

void cpu_version(int* value, int value_extent, float* mopr, float* Mopr){
    struct timespec initial, finish;
    clock_gettime(CLOCK_MONOTONIC, &initial);
    omp_set_num_threads(omp_threads_number);
    int i, j;
    int sequence_threads, sequence_bins, threads_number;
    int* bin_init, *bin_init_aft;
    bin_init = (int *)calloc(omp_threads_number * bins_number, sizeof(int));
    bin_init_aft = (int *)calloc(bins_number, sizeof(int));

    #pragma omp parallel for private(i, sequence_threads)
    for(i = 0; i < value_extent; i++){
        sequence_threads = omp_get_num_threads();
        if(sequence_threads == 0){
            threads_number = omp_get_num_threads();
        }
        sequence_bins = bins_verify(value[i], mopr, Mopr);
        assert(("class index should not be negative", sequence_bins >= 0));
        #pragma omp atomic
        bin_init[sequence_threads*bins_number + sequence_bins] += 1;
    }
    int tmp = 0;
    #pragma omp parallel for schedule(static) private(i, j) reduction(+:tmp)
    for(i = 0; i < bins_number; i++){
        tmp = 0;
        for(j = 0; j < omp_threads_number; j++){
            tmp += bin_init[j * bins_number + i];
        }
        bin_init_aft[i] = tmp;
    }
    clock_gettime(CLOCK_MONOTONIC, &finish);
    double duration = (finish.tv_sec - initial.tv_sec);
    duration += (finish.tv_nsec - initial.tv_nsec) / 1000000000.0;
    printf("CPU Time: %f\n", duration);
    //hist_output(bin_init_aft);
}

int main(int argc, char *argv[]){
    assert(("./Question1 <number of values>", argc == 2));
    int value_extent = atoi(argv[1]);
    int * value; 
    float * mopr, * Mopr;
    value = (int *)calloc(value_extent, sizeof(int));
    mopr = (float *)calloc(bins_number, sizeof(float));
    Mopr = (float *)calloc(bins_number, sizeof(float));

    value_initialization(value, value_extent);
    bins_installing(mopr, Mopr, bins_number);
    cuda_version(value, value_extent, mopr, Mopr);
    cpu_version(value, value_extent, mopr, Mopr);
    
    return 0;
}